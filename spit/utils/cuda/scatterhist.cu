
#include <hip/hip_runtime.h>
extern "C" __global__ void flatnorm_scatterhist_kernel(
    const float* features, 
    float* output, 
    const long long* indices, 
    const float* bins, 
    const float* sigmaptr, 
    const long long num_pixels, 
    const long long num_features, 
    const long long num_bins
) {

    long long thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long thread_cnt = gridDim.x * blockDim.x;
    const float sigma = sigmaptr[0];

    for (long long pixel_idx = thread_idx; pixel_idx < num_pixels; pixel_idx += thread_cnt) {
        long long output_idx_base = indices[pixel_idx] * num_bins * num_features;

        for (long long feature = 0; feature < num_features; ++feature) {
            float feature_val = features[pixel_idx * num_features + feature];
            for (long long bin = 0; bin < num_bins; ++bin) {
                float bin_val = bins[bin];
                float z = (feature_val - bin_val) / sigma;
                float hist_val = exp(-0.5 * z * z);
                
                // Calculate the output index
                long long output_idx = output_idx_base + feature * num_bins + bin;

                // Atomic add to the output
                atomicAdd(&output[output_idx], hist_val);
            }
        }
    }
}

extern "C" __global__ 
void scatter_joint_hist(
    const long long* seg,
    const float* feats,
    const float* mesh_y,
    const float* mesh_x,
    const long long* featcombs,
    float* output,
    float* sigmaptr,
    const long long n,
    const long long nbins,
    const long long nfeats,
    const long long feat_dim
) {
    long long idx = blockDim.x * blockIdx.x + threadIdx.x;
    long long nbins2 = nbins * nbins;
    float sigma = sigmaptr[0];
        
    if (idx < n) {
        long long s_idx = seg[idx];
        float y;
        float x;
        float z1;
        float z2;
        float value;
        long long j_y;
        long long j_x;
        
        for (long long j = 0; j < nfeats; j++){
            j_y = featcombs[2*j];
            j_x = featcombs[2*j+1];
            y = feats[idx*feat_dim + j_y];
            x = feats[idx*feat_dim + j_x];
            
            for (long long i = 0; i < nbins2; i++) {
                z1 = (y - mesh_y[i]) / sigma;
                z2 = (x - mesh_x[i]) / sigma;
                value = exp(-0.5 * (z1 * z1 + z2 * z2));
                atomicAdd(&output[s_idx * nfeats * nbins2 + j * nbins2 + i], value);
            }
        }
    }
}

